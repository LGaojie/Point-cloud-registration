#include "hip/hip_runtime.h"
#include <stdio.h>
#include "RegAlgrm_hxmc.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#define BLOCK_SIZE 512

char szErrMsg[512];
texture<float, 1, hipReadModeElementType> texA1D;
texture<float, 1, hipReadModeElementType> texB1D;

/*******************************************************************************
˵����
	���cuda�Ĵ�����Ϣ������ȫ�ֱ������GetErrorMsg�õ�������Ϣ��
������
	error�� cuda�Ĵ�����Ϣ����
 ********************************************************************************/
void PrintErrorMessage(hipError_t error)
{
	sprintf(szErrMsg, "cuda error: %s\n.", hipGetErrorString(error));
}

/*******************************************************************************
˵����
	���cuda�Ĵ�����Ϣ��
������
	�ޡ�
���أ�
	������Ϣ���ַ�����
 ********************************************************************************/
char* GetErrorMsg()
{
	return szErrMsg;
}

/*******************************************************************************
˵����
	�˺�����������GPU�С�
	���������ڴ棬����㼯A(�ο�)��㼯B(��ѯ)��֮����Եľ��루B��ÿ�㵽A��ÿһ�㣩��
	��Ҫ16 * (nAPtNum%16+1) * 16 * (nBPtNum%16 + 1)���̡߳�
ǰ�᣺
	�㼯A�Ѿ��洢��һ�������õ�GPU������buffer��texA1D�У�
	�㼯B�Ѿ��洢��һ�������õ�GPU������buffer��texB1D�У�

������
	nAPtNum��    �㼯A�еĵ���(reference)��
	nBPtNum��    �㼯B�еĵ���(query)��
	nDistPitch�� �㼯�����У��������ļ�ࣨ�ֽڣ���
	pfDistBuf��  ָ���������nAPtNum * nBPtNum������������
 ********************************************************************************/
__global__ void cuComputeDistanceTexture1D(int nAPtNum, int nBPtNum, int nDistPitch, float* pfDistBuf)
{
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if (xIndex < nAPtNum && yIndex < nBPtNum)
	{
		float fSqSum = 0;
		for (int i = 0; i < 3; i++)
		{
			float tmp = tex1Dfetch(texA1D, i * nAPtNum + xIndex) - tex1Dfetch(texB1D, i * nBPtNum + yIndex);
			fSqSum += tmp * tmp;
		}
		fSqSum = sqrt(fSqSum);
		pfDistBuf[yIndex * nDistPitch + xIndex] = fSqSum;
	}
}

/*******************************************************************************
˵����
	�˺�����������GPU�С�
	����B�㼯�����ڽ��㡣
ǰ�᣺
	�㼯A�Ѿ��洢��һ�������õ�GPU������buffer��texA1D�У�
	�㼯B��ÿһ�㵽�㼯A��ÿһ��ľ����Ѿ�����ã�����GPUȫ���ڴ�pfDist_dev�У�

������
	nAPtNum��    �㼯A�еĵ���(reference)��
	nBPtNum��    �㼯B�еĵ���(query)��
	nDistPitch�� �㼯�����У��������ļ�ࣨ�ֽڣ���
	pfDist_dev�� �㼯B��ÿһ�㵽�㼯A��ÿһ��ľ��룻
	pnDistIdx��  ��A����������ָ����
	pfNearDist�� ��A���������ľ��롣
 ********************************************************************************/
__global__ void cuGetNearestPt(int nAPtNum, int nBPtNum, int nDistPitch, float* pfDist_dev, int* pnDistIdx, float* pfNearDist)
{
	float  fDistT;
	float  fDis = 1.0E10;
	int    nIndex;
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (xIndex < nBPtNum)
	{

		for (int i = 0; i < nAPtNum; i++)
		{
			fDistT = pfDist_dev[xIndex * xIndex + i];
			if (fDistT < fDis)
			{
				fDis = fDistT;
				nIndex = i;
			}
		}

		pnDistIdx[xIndex] = nIndex;
		pfNearDist[xIndex] = fDis;
	}
}

/*******************************************************************************
˵����
	�˺�����������GPU�С�
	����pfDist��ָ�򸡵������У�����ֿ�(BLOCK_SIZE��С�Ŀ�)����ÿ�������С�ĵ��Լ�λ�á�

������
	���룺
		pfDist��      �����device��Bufferָ�룻
		nDataNum��    �㼯�и�������������
	�����
		pfOutBlock��  ÿ������С�ľ���buffer��
		pnIndexBlock��ÿ������С�ľ����Ӧ�ڵ㼯�еı�ţ�
 ********************************************************************************/
__global__ void cuFindNearestPt(float* pfDist, int nDataNum, float* pfOutBlock, int* pnIndexBlock)
{
	__shared__ float sfData[BLOCK_SIZE];
	__shared__ int   snData[BLOCK_SIZE];

	unsigned int tid = threadIdx.x;
	unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

	// load shared mem
	sfData[tid] = (gid < nDataNum) ? pfDist[gid] : 0;
	snData[tid] = tid;

	// do reduction in shared mem, stride is divided by 2,
	for (unsigned int nHalfB = blockDim.x / 2; nHalfB > 0; nHalfB >>= 1)
	{
		__syncthreads();

		if (tid < nHalfB)
		{
			if (sfData[tid] > sfData[tid + nHalfB])
			{
				sfData[tid] = sfData[tid + nHalfB];
				snData[tid] = tid + nHalfB;
			}
		}
	}

	// write result for this block to global mem
	if (tid == 0)
	{
		pfOutBlock[blockIdx.x] = sfData[0];
		pnIndexBlock[blockIdx.x] = snData[0];
	}
}

/*****************************************************************************************************************
˵����
	�˺�����������GPU�С�
	�����ѯ����pfQueSet��ÿ�㵽�ο�����pfRefSet��������룬�Լ���Ӧ������ڲο��㼯�ı�š�
������
	���룺
		pfRefSet�� ���ռ�3D���ο��㼯��HOSTָ�룻
		nRefNum��   �ο����������
		pfQueSet��  ��ѯ�㼯��HOSTָ�룻
		nQueNum��   ��ѯ���������
	�����
		fpDist��    ���ѯ�㼯��Ӧ��ŵ���С���룻
		pnIndex��   ��Ӧ�ο��㼯�ı�ţ���0��ʼ����
 ***************************************************************************************************************** */
__global__ void cuKnn1_hxmc(float* pfRefSet, int nRefNum, float* pfQueSet, int nQueNum, float* fpDist, int* pnIndex)
{
	__shared__ float sfData[BLOCK_SIZE + 1];
	__shared__ int   snData[BLOCK_SIZE + 1];
  
	unsigned int tid = threadIdx.x;
	unsigned int gid;
	int          nCircN = (nRefNum + BLOCK_SIZE - 1) / BLOCK_SIZE;
	float tmp1 = tex1Dfetch(texB1D, blockIdx.x);
	float tmp2 = tex1Dfetch(texB1D, blockIdx.x + nQueNum);
	float tmp3 = tex1Dfetch(texB1D, blockIdx.x + 2 * nQueNum);

	if (tid == 0)
	{
		sfData[BLOCK_SIZE] = 1.0E10;
	}

	for (int i = 0; i < nCircN; i++)      //ÿ��block����һ����ѯ�㣬��Ҫѭ��nCircN���ܼ���������ο��㼯�ľ���
	{
		//1. ���㱾��ѭ���Ĳ�ѯ����BLOCK_SIZE���ο���ľ���
		gid = BLOCK_SIZE * i + tid;       //�ο��㼯��λ��

		if (gid < nRefNum)
		{
			float tmp = (tmp1 - tex1Dfetch(texA1D, gid)) * (tmp1 - tex1Dfetch(texA1D, gid));
			tmp += (tmp2 - tex1Dfetch(texA1D, gid + nRefNum)) * (tmp2 - tex1Dfetch(texA1D, gid + nRefNum));
			tmp += (tmp3 - tex1Dfetch(texA1D, gid + 2 * nRefNum)) * (tmp3 - tex1Dfetch(texA1D, gid + 2 * nRefNum));
			sfData[tid] = sqrt(tmp);
			snData[tid] = gid;
		}
		else
		{
			sfData[tid] = 1.0E10;
			snData[tid] = -1;
		}
		__syncthreads();

		//2. ���㱾���е���С����Ͳο��㼯λ��
		for (unsigned int nHalfB = blockDim.x / 2; nHalfB > 0; nHalfB >>= 1)
		{
			__syncthreads();

			if (tid < nHalfB)
			{
				if (sfData[tid] > sfData[tid + nHalfB])
				{
					sfData[tid] = sfData[tid + nHalfB];
					snData[tid] = snData[tid + nHalfB];
				}
			}
		}

		//3. write result for this circle to shared mem.
		if (tid == 0)
		{
			if (sfData[BLOCK_SIZE] > sfData[0])
			{
				sfData[BLOCK_SIZE] = sfData[0];
				snData[BLOCK_SIZE] = snData[0];
			}
		}
	} //for (int i = 0; i < nCircN; i++)

	if (tid == 0)
	{
		fpDist[blockIdx.x] = sfData[BLOCK_SIZE];
		pnIndex[blockIdx.x] = snData[BLOCK_SIZE];
	}
}

/*****************************************************************************************************************
˵����
	ʵ��nRefNum�����Դ����ÿ���㵽nQueNum��Ŀ����Ƶľ�����㣬�������е㵽��ľ��롣
������
	pfRefSet�� ���ռ�3D���ο��㼯��HOSTָ�룻
	nRefNum��   �ο����������
	pfQueSet��  ��ѯ�㼯��HOSTָ�룻
	nQueNum��   ��ѯ���������
	fpDistance��������ĵ㵽������HOST���棨��pfQueSet*pfRefSet����������pfRefSet�У�pfQueSet�У���
���أ�
	�ɹ����㽫���ؾ������������Կ��ڴ治��������-1��
 ***************************************************************************************************************** */
int PointSetDist_hxmc(float* pfRefSet, int nRefNum, float* pfQueSet, int nQueNum, float* fpDistance)
{
	float* pfRefSet_dev;
	float* pfQueSet_dev;
	float* pfDist_dev;
	size_t       nDistPitch;
	size_t       nDistPitchByte;
	hipError_t  cuErr;

	//�����ڴ���ο��Ͳ�ѯ��
	cuErr = hipMalloc((void**)&pfRefSet_dev, nRefNum * sizeof(float) * 3);
	if (cuErr)
	{
		PrintErrorMessage(cuErr);
		return -1;
	}

	cuErr = hipMalloc((void**)&pfQueSet_dev, nQueNum * sizeof(float) * 3);
	if (cuErr)
	{
		hipFree(pfRefSet_dev);
		PrintErrorMessage(cuErr);
		return -2;
	}

	hipMemcpy(pfRefSet_dev, pfRefSet, nRefNum * sizeof(float) * 3, hipMemcpyHostToDevice);
	hipMemcpy(pfQueSet_dev, pfQueSet, nQueNum * sizeof(float) * 3, hipMemcpyHostToDevice);

	hipBindTexture(0, texA1D, pfRefSet_dev);
	hipBindTexture(0, texB1D, pfQueSet_dev);

	//�����ڴ��������
	// Allocation of global memory for distance buffer.	
	cuErr = hipMallocPitch((void**)&pfDist_dev, &nDistPitchByte, nRefNum * sizeof(float), nQueNum);
	if (cuErr)
	{
		hipFree(pfRefSet_dev);
		hipFree(pfQueSet_dev);
		PrintErrorMessage(cuErr);

		return -3;
	}
	nDistPitch = nDistPitchByte / sizeof(float);

	// Grids ans Blocks
	dim3 dmGrad_16x16(nQueNum / 16, nRefNum / 16, 1);
	dim3 dmBlock_16x16(16, 16, 1);

	if (nQueNum % 16 != 0)
	{
		dmGrad_16x16.x += 1;
	}
	if (nRefNum % 16 != 0)
	{
		dmGrad_16x16.y += 1;
	}

	cuComputeDistanceTexture1D << <dmGrad_16x16, dmBlock_16x16 >> > (nRefNum, nQueNum, nDistPitch, pfDist_dev);

	//// Memory copy of output from device to host
	hipMemcpy2D(fpDistance, nRefNum * sizeof(float), pfDist_dev, nDistPitchByte, nRefNum * sizeof(float), nQueNum, hipMemcpyDeviceToHost);

	// Free memory
	hipUnbindTexture(texA1D);
	hipUnbindTexture(texB1D);
	hipFree(pfRefSet_dev);
	hipFree(pfQueSet_dev);
	hipFree(pfDist_dev);

	return nRefNum * nQueNum;
}

/*****************************************************************************************************************
˵����
	�ҵ��㼯pfQueSet��ÿ�㵽�ο��㼯pfRefSet������㣬���������ľ���ͱ�ţ�0��׼����
������
	���룺
	pfRefSet���ο��㼯��HOSTָ�룻
	nRefNum�� �ο����������
	pfQueSet����ѯ�㼯��HOSTָ�룻
	nQueNum�� ��ѯ���������
	�����
	pfDistBuf(out)��  pfQueSetÿ���㵽pfRefSet�㼯�����ľ��룻
	pnRstIdxBuf(out)����ӦpfDistBuf�ĵ��ڲο�����pfRefSet��������0��׼����
 ******************************************************************************************************************/
extern "C" __declspec(dllexport)  int GetNearestPt_hxmc(
	float* pfRefSet, int nRefNum,
	float* pfQueSet, int nQueNum,
	float* pfNearDist,
	int* pnRstIdxBuf)
{
	float* pfRefSet_dev;
	float* pfQueSet_dev;
	float* pfDist_dev;
	float* pfNearDist_dev;
	int*   pnDistIdx_dev;
	size_t       nDistPitch;
	size_t       nDistPitchByte;
	hipError_t  cuErr;

	//�����ڴ���ο��Ͳ�ѯ��
	cuErr = hipMalloc((void**)&pfRefSet_dev, nRefNum * sizeof(float) * 3);
	if (cuErr)
	{
		PrintErrorMessage(cuErr);
		return -1;
	}

	cuErr = hipMalloc((void**)&pfQueSet_dev, nQueNum * sizeof(float) * 3);
	if (cuErr)
	{
		hipFree(pfRefSet_dev);
		PrintErrorMessage(cuErr);
		return -2;
	}

	hipMemcpy(pfRefSet_dev, pfRefSet, nRefNum * sizeof(float) * 3, hipMemcpyHostToDevice);
	hipMemcpy(pfQueSet_dev, pfQueSet, nQueNum * sizeof(float) * 3, hipMemcpyHostToDevice);

	hipBindTexture(0, texA1D, pfRefSet_dev);
	hipBindTexture(0, texB1D, pfQueSet_dev);

	//�����ڴ��������
	// Allocation of global memory for distance buffer.	
	cuErr = hipMallocPitch((void**)&pfDist_dev, &nDistPitchByte, nRefNum * sizeof(float), nQueNum);
	if (cuErr)
	{
		hipFree(pfRefSet_dev);
		hipFree(pfQueSet_dev);
		PrintErrorMessage(cuErr);

		return -3;
	}
	nDistPitch = nDistPitchByte / sizeof(float);

	// Grids ans Blocks
	dim3 dmGrad_16x16(nQueNum / 16, nRefNum / 16, 1);
	dim3 dmBlock_16x16(16, 16, 1);

	if (nQueNum % 16 != 0)
	{
		dmGrad_16x16.x += 1;
	}
	if (nRefNum % 16 != 0)
	{
		dmGrad_16x16.y += 1;
	}

	cuComputeDistanceTexture1D << <dmGrad_16x16, dmBlock_16x16 >> > (nRefNum, nQueNum, nDistPitch, pfDist_dev);

	// Free memory
	hipUnbindTexture(texA1D);
	hipUnbindTexture(texB1D);
	hipFree(pfRefSet_dev);
	hipFree(pfQueSet_dev);

	cuErr = hipMalloc((void**)&pfNearDist_dev, nQueNum * sizeof(float));
	if (cuErr)
	{
		hipFree(pfDist_dev);
		PrintErrorMessage(cuErr);
		return -4;
	}

	cuErr = hipMalloc((void**)&pnDistIdx_dev, nQueNum * sizeof(int));
	if (cuErr)
	{
		hipFree(pfDist_dev);
		hipFree(pfNearDist_dev);
		PrintErrorMessage(cuErr);
		return -5;
	}

	dim3 dmGrad_512(nQueNum / 512, 1, 1);
	dim3 dmBlock_512(512, 1, 1);
	cuGetNearestPt << <dmGrad_512, dmBlock_512 >> > (nRefNum, nQueNum, nDistPitch, pfDist_dev, pnDistIdx_dev, pfNearDist_dev);

	hipMemcpy(pnRstIdxBuf, pnDistIdx_dev, nQueNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(pfNearDist, pfNearDist_dev, nQueNum * sizeof(float), hipMemcpyDeviceToHost);

	// Free memory
	hipFree(pfDist_dev);
	hipFree(pfNearDist_dev);
	hipFree(pnDistIdx_dev);

	return 0;
}

/*******************************************************************************************
˵����
	ʵ��nRefNum���㼯�У�ά��nPtDim����nQueNum���������ٵ�1�������
������
	���룺
		pfRefSet���ο��㼯��HOSTָ�룻
		nRefNum�� �ο����������
		pfQueSet����ѯ�㼯��HOSTָ�룻
		nQueNum�� ��ѯ���������
	�����
		pfDistBuf(out)��  pfQueSetÿ�������k����ľ����Buffer��k*nQueNum������������
		pnRstIdxBuf(out)����ӦpfDistBuf�ĵ��ڲο����Ƶ�������
 *******************************************************************************************/
int Knn1_hxmc(float* pfRefSet, int nRefNum, float* pfQueSet, int nQueNum, float* pfDistBuf, int* pnRstIdxBuf)
{
	dim3   dimGrid(nQueNum, 1, 1);
	dim3   dimBlock(BLOCK_SIZE, 1, 1);

	float* pfRefSet_d;
	float* pfQueSet_d;
	float* pfDistBuf_d;
	int*   pnRstIdxBuf_d;
	hipError_t statCuda;

	statCuda = hipMalloc(reinterpret_cast<void**>(&pfRefSet_d), sizeof(float) * nRefNum *3);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -1;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&pfQueSet_d), sizeof(float) * nQueNum * 3);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -2;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&pfDistBuf_d), sizeof(float) * nQueNum);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -3;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&pnRstIdxBuf_d), sizeof(int) * nQueNum);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -4;
	}

	statCuda = hipMemcpy(pfRefSet_d, pfRefSet, sizeof(float) * nRefNum * 3, hipMemcpyHostToDevice);
	statCuda = hipMemcpy(pfQueSet_d, pfQueSet, sizeof(float) * nQueNum * 3, hipMemcpyHostToDevice);

	hipBindTexture(0, texA1D, pfRefSet_d);
	hipBindTexture(0, texB1D, pfQueSet_d);

	cuKnn1_hxmc<<<dimGrid , dimBlock >>>(pfRefSet_d, nRefNum, pfQueSet_d, nQueNum, pfDistBuf_d, pnRstIdxBuf_d);

	statCuda = hipMemcpy(pfDistBuf, pfDistBuf_d, sizeof(float) * nQueNum, hipMemcpyDeviceToHost);
	statCuda = hipMemcpy(pnRstIdxBuf, pnRstIdxBuf_d, sizeof(float) * nQueNum, hipMemcpyDeviceToHost);

	hipBindTexture(0, texA1D, pfRefSet_d);
	hipBindTexture(0, texB1D, pfQueSet_d);

	statCuda = hipFree(pfRefSet_d);
	statCuda = hipFree(pfQueSet_d);
	statCuda = hipFree(pfDistBuf_d);
	statCuda = hipFree(pnRstIdxBuf_d);

	return 0;
}

/*******************************************************************************
˵����
	����pfDist��ָ�򸡵������У�������С�ĵ��Լ�λ�á�
������
	���룺
		pfDist��      �������������룩Bufferָ�룻
		nDataNum��    �㼯�и�������������
	�����
		fNearestDist��������С�ĸ�������
		pnIndexBlock����С����Ӧ�ڵ㼯�еı�ţ�
 ********************************************************************************/
int FindNearestPt(float* pfDist, unsigned int nDataNum, float& fNearestDist, int& nIndex)
{
	int    nBlockNum = (nDataNum + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3   dimGrid(nBlockNum, 1, 1);
	dim3   dimBlock(BLOCK_SIZE, 1, 1);
	float* pfDist_d;
	float* pfNearBlock_d;
	int*   pnNearInxBlk_d;
	hipError_t statCuda;

	statCuda = hipMalloc(reinterpret_cast<void**>(&pfDist_d), sizeof(float) * nDataNum);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -1;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&pfNearBlock_d), sizeof(float) * nBlockNum);
	if (statCuda != hipSuccess)
	{
		statCuda = hipFree(pfDist_d);
		PrintErrorMessage(statCuda);
		return -2;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&pnNearInxBlk_d), sizeof(int) * nBlockNum);
	if (statCuda != hipSuccess)
	{
		statCuda = hipFree(pfDist_d);
		statCuda = hipFree(pfNearBlock_d);
		PrintErrorMessage(statCuda);
		return -3;
	}

	statCuda = hipMemcpy(pfDist_d, pfDist, sizeof(float) * nDataNum, hipMemcpyHostToDevice);
	if (statCuda != hipSuccess)
	{
		statCuda = hipFree(pfDist_d);
		statCuda = hipFree(pfNearBlock_d);
		statCuda = hipFree(pnNearInxBlk_d);

		PrintErrorMessage(statCuda);
		return -4;
	}

	//��һ�β��ҷֿ�ļ��ϸ��Ե���С��
	cuFindNearestPt << <dimGrid, dimBlock >> > (pfDist_d, nDataNum, pfNearBlock_d, pnNearInxBlk_d);

	if (nBlockNum == 1)
	{
		statCuda = hipMemcpy(&fNearestDist, pfNearBlock_d, sizeof(float), hipMemcpyDeviceToHost);
		statCuda = hipMemcpy(&nIndex, pnNearInxBlk_d, sizeof(int), hipMemcpyDeviceToHost);

		statCuda = hipFree(pfDist_d);
		statCuda = hipFree(pfNearBlock_d);
		statCuda = hipFree(pnNearInxBlk_d);

		return 0;
	}

	//�ڵ�һ�β��ҷ��صĸ����������У��ڶ��β�����С��
	int    nBlockNum2 = (nBlockNum + BLOCK_SIZE - 1) / BLOCK_SIZE;
	float* pfNearBlock2_d;
	int* pnNearInxBlk2_d;
	int* pnNearInxBlk = new int[nBlockNum];

	dimGrid.x = nBlockNum2;
	statCuda = hipMemcpy(pnNearInxBlk, pnNearInxBlk_d, sizeof(int) * nBlockNum, hipMemcpyDeviceToHost);

	statCuda = hipMalloc(reinterpret_cast<void**>(&pfNearBlock2_d), sizeof(float) * nBlockNum2);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -1;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&pnNearInxBlk2_d), sizeof(int) * nBlockNum2);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -2;
	}

	cuFindNearestPt << <dimGrid, dimBlock >> > (pfNearBlock_d, nBlockNum, pfNearBlock2_d, pnNearInxBlk2_d);

	if (nBlockNum2 == 1)
	{
		int  nIndex2;

		statCuda = hipMemcpy(&fNearestDist, pfNearBlock_d, sizeof(float), hipMemcpyDeviceToHost);
		statCuda = hipMemcpy(&nIndex2, pnNearInxBlk_d, sizeof(int), hipMemcpyDeviceToHost);

		nIndex = pnNearInxBlk[nIndex2];

		statCuda = hipFree(pfDist_d);
		statCuda = hipFree(pfNearBlock_d);
		statCuda = hipFree(pnNearInxBlk_d);
		statCuda = hipFree(pfNearBlock2_d);
		statCuda = hipFree(pnNearInxBlk2_d);

		delete[] pnNearInxBlk;

		return 0;
	}
	else
	{
		statCuda = hipFree(pfDist_d);
		statCuda = hipFree(pfNearBlock_d);
		statCuda = hipFree(pnNearInxBlk_d);
		statCuda = hipFree(pfNearBlock2_d);
		statCuda = hipFree(pnNearInxBlk2_d);

		delete[] pnNearInxBlk;

		return -6;   //֧�����512*512������������-6
	}

	//�ڵڶ��β��ҷ��صĸ����������У������β�����С������������Բ��ң�BLOCK_SIZE*BLOCK_SIZE*BLOCK_SIZE������������С����
	//�´�������

	return 0;
}

/*************************************************************************
˵����
	��3x3��˫���Ⱦ������SVD�ֽ⡣����㷨����cuda��cuSolver��ʵ�֡�

	ע�⣺cuda�������������ȵģ���ˣ����ݵĲ����ǣ�
	(1,1), (2,1),(3,1);(1,2), (2,2),(3,2);(1,3), (2,3),(3,3)��˳��
������
	���룺
		pdA�����ֽ��3x3˫����ʵ������
	�����
		pdU�� 3x3˫����ʵ������U��
		pdS�� �ԽǾ���ĶԽ��ߣ��Ӵ�С���У�
		pdVT��3x3˫����ʵ������V��ת�á�
���أ�
	�ɹ�����0��ʧ�ܷ���һ���������ɵ���GetErrorMsg�����õ��������Ϣ��
 **************************************************************************/
int SVD3x3_hxmc(double* pdA, double* pdU, double* pdS, double* pdVT)
{
	hipsolverHandle_t cusolverH = NULL;
	//hipblasHandle_t     cublasH = NULL;
	hipStream_t       stream = NULL;
	hipsolverStatus_t   statCusolver;
	//hipblasStatus_t     statCublas;
	hipError_t        statCuda;

	const int m = 3;       // 3*3 Matrix
	const int n = 3;       // 3*3 Matrix
	const int lda = m;     // lda = m����������к���

	int info_gpu = 0;      // host copy of error info

	double* d_A = nullptr;
	double* d_S = nullptr;  // singular values
	double* d_U = nullptr;  // left singular vectors
	double* d_VT = nullptr; // right singular vectors
	int*    devInfo = nullptr;
	int     lwork = 0;          // size of workspace
	double* d_work = nullptr;
	double* d_rwork = nullptr;

	// step 1: create cusolver handle and cublas handle
	statCusolver = hipsolverDnCreate(&cusolverH);
	if (statCusolver != HIPSOLVER_STATUS_SUCCESS)
	{
		sprintf(szErrMsg, "cuSolver error: Create Handle.\n.");
		return -1;
	}

	/*statCublas = hipblasCreate(&cublasH);
	if (statCublas != HIPBLAS_STATUS_SUCCESS)
	{
		sprintf(szErrMsg, "cuBlas error: Create Handle.\n.");
		return -2;
	}*/

	statCuda = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	statCusolver = hipsolverSetStream(cusolverH, stream);

	// step 2: copy A to device
	statCuda = hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(double) * 9);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -3;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&d_S), sizeof(double) * 3);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -4;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&d_U), sizeof(double) * 9);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -5;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&d_VT), sizeof(double) * 9);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -6;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&devInfo), sizeof(int));
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -7;
	}

	statCuda = hipMemcpyAsync(d_A, pdA, sizeof(double) * 9, hipMemcpyHostToDevice, stream);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -8;
	}

	// step 3: query working space of SVD
	statCusolver = hipsolverDnDgesvd_bufferSize(cusolverH, m, n, &lwork);
	if (statCusolver != HIPSOLVER_STATUS_SUCCESS)
	{
		sprintf(szErrMsg, "cuSolver error: Query workspace.\n.");
		return -9;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&d_work), sizeof(double) * lwork);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -10;
	}

	// step 4: compute SVD
	signed char jobu = 'A';  // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	statCusolver = hipsolverDnDgesvd(cusolverH, jobu, jobvt, m, n, d_A, lda, d_S, d_U,
		lda, // ldu
		d_VT,
		lda, // ldvt,
		d_work, lwork, d_rwork, devInfo);
	if (statCusolver != HIPSOLVER_STATUS_SUCCESS)
	{
		sprintf(szErrMsg, "cuSolver error: Solver matrix (svd).\n.");
		return -11;
	}

	//step5: copy data to host
	statCuda = hipMemcpyAsync(pdU, d_U, sizeof(double) * 9, hipMemcpyDeviceToHost, stream);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -12;
	}

	statCuda = hipMemcpyAsync(pdVT, d_VT, sizeof(double) * 9, hipMemcpyDeviceToHost,	stream);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -13;
	}

	statCuda = hipMemcpyAsync(pdS, d_S, sizeof(double) * 3, hipMemcpyDeviceToHost, stream);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -14;
	}

	statCuda = hipMemcpyAsync(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost, stream);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -15;
	}

	statCuda = hipStreamSynchronize(stream);

	//step6: free resources
	statCuda =hipFree(d_A);
	statCuda =hipFree(d_U);
	statCuda =hipFree(d_VT);
	statCuda =hipFree(d_S);
	statCuda =hipFree(devInfo);
	statCuda =hipFree(d_work);
	statCuda =hipFree(d_rwork);

	statCusolver = hipsolverDnDestroy(cusolverH);

	statCuda = hipStreamDestroy(stream);
	statCuda = hipDeviceReset();

	return 0;
}

/*************************************************************************
˵����
	���6x6���Է����飺Ax = b��˫�������㡣����㷨����cuda��cuSolver��LU�ֽ�ʵ�֡�

	ע�⣺cuda�������������ȵģ���ˣ�A���󴫵ݲ�����˳���ǣ�
	(1,1), (2,1),(3,1)��(4,1), (5,1),(6,1)��
	(1,2), (2,2),(3,2)��(4,2), (5,2),(6,2)��
	(1,3), (2,3),(3,3)��(4,3), (5,3),(6,3)��
	(1,4), (2,4),(3,4)��(4,4), (5,4),(6,4)��
	(1,5), (2,5),(3,5)��(4,5), (5,5),(6,5)��
	(1,6), (2,6),(3,6)��(4,6), (5,6),(6,6)��
������
	���룺
		pdA�����̵�ϵ������
		pdB�����̵ĳ���ʸ����
	�����
		pdX�� ��ʸ����
���أ�
	�ɹ�����0��ʧ�ܷ���һ���������ɵ���GetErrorMsg�����õ��������Ϣ��
 **************************************************************************/
int Solver6x6Equ_hxmc(double* pdA, double* pdB, double* pdX)
{
	hipsolverHandle_t cusolverH = NULL;
	hipStream_t       stream = NULL;
	hipError_t        statCuda;
	hipsolverStatus_t   statCusolver;

	//host
	const int m = 6;
	const int lda = m;
	const int ldb = m;
	double* pdLU = new double[lda * m];
	int*    pnIpiv = new int[m];
	int     nInfo = 0;
	//device
	double* pdA_d = nullptr;        // device copy of A
	double* pdB_d = nullptr;        // device copy of B
	int*    pnIpiv_d = nullptr;     // pivoting sequence
	int*    pnInfo_d = nullptr;     // error info
	int     lwork = 0;              // size of workspace
	double* pdWork_d = nullptr;     // device workspace for getrf

	memset((void*)pnIpiv, 0, m * sizeof(int));
	memset((void*)pdLU, 0, lda * m * sizeof(double));
	memset((void*)pdX, 0, m * sizeof(double));

	// step 1: create cusolver handle, bind a stream
	statCusolver = hipsolverDnCreate(&cusolverH);
	if (statCusolver != HIPSOLVER_STATUS_SUCCESS)
	{
		sprintf(szErrMsg, "cuSolver error: Create Handle.\n.");
		return -1;
	}

	statCuda = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	if (statCuda != hipSuccess)
	{
		PrintErrorMessage(statCuda);
		return -2;
	}

	statCusolver = hipsolverSetStream(cusolverH, stream);
	if (statCusolver != HIPSOLVER_STATUS_SUCCESS)
	{
		sprintf(szErrMsg, "cuSolver error: Set cudaStream.\n.");
		return -3;
	}

	/* step 2: copy A to device */
	statCuda = hipMalloc(reinterpret_cast<void**>(&pdA_d), sizeof(double) * m *m );
	statCuda = hipMalloc(reinterpret_cast<void**>(&pdB_d), sizeof(double) * m);
	statCuda = hipMalloc(reinterpret_cast<void**>(&pnIpiv_d), sizeof(int) * m);
	statCuda = hipMalloc(reinterpret_cast<void**>(&pnInfo_d), sizeof(int));

	statCuda = hipMemcpyAsync(pdA_d, pdA, sizeof(double) * m * m, hipMemcpyHostToDevice, stream);
	statCuda = hipMemcpyAsync(pdB_d, pdB, sizeof(double) * m, hipMemcpyHostToDevice, stream);

	// step 3: query working space of getrf
	statCusolver = hipsolverDnDgetrf_bufferSize(cusolverH, m, m, pdA_d, lda, &lwork);
	if (statCusolver != HIPSOLVER_STATUS_SUCCESS)
	{
		sprintf(szErrMsg, "cuSolver error: Query workspace.\n.");

		delete[] pdLU;
		delete[] pnIpiv;

		// free resources
		statCuda = hipFree(pdA_d);
		statCuda = hipFree(pdB_d);
		statCuda = hipFree(pnIpiv_d);
		statCuda = hipFree(pnInfo_d);

		return -4;
	}

	statCuda = hipMalloc(reinterpret_cast<void**>(&pdWork_d), sizeof(double) * lwork);

	// step 4: LU factorization
	statCusolver = hipsolverDnDgetrf(cusolverH, m, m, pdA_d, lda, pdWork_d, pnIpiv_d, pnInfo_d);

	// ȡ��LU��� Test code
	statCuda = hipMemcpyAsync(pnIpiv, pnIpiv_d, sizeof(int) * m, hipMemcpyDeviceToHost, stream);        //��Ԫ��λ��
	statCuda = hipMemcpyAsync(pdLU, pdA_d, sizeof(double) * m * m, hipMemcpyDeviceToHost, stream);      //LU���Ǿ���;
	statCuda = hipMemcpyAsync(&nInfo, pnInfo_d, sizeof(int), hipMemcpyDeviceToHost, stream);            //�����Ϣ
	statCuda = hipStreamSynchronize(stream);

	if (0 > nInfo)
	{
		sprintf(szErrMsg, "cuSolver error: LU factorization %d-th Parameter is wrong.\n.", -nInfo);

		delete[] pdLU;
		delete[] pnIpiv;

		// free resources
		statCuda = hipFree(pdA_d);
		statCuda = hipFree(pdB_d);
		statCuda = hipFree(pnIpiv_d);
		statCuda = hipFree(pnInfo_d);
		statCuda = hipFree(pdWork_d);
		return -5;
	}

	// step 5: solve A*X = B
	statCusolver = hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, m, 1, pdA_d, lda, pnIpiv_d, pdB_d, ldb, pnInfo_d);
	if (statCusolver != HIPSOLVER_STATUS_SUCCESS)
	{
		sprintf(szErrMsg, "cuSolver error: Solve equation.\n.");

		delete[] pdLU;
		delete[] pnIpiv;

		// free resources
		statCuda = hipFree(pdA_d);
		statCuda = hipFree(pdB_d);
		statCuda = hipFree(pnIpiv_d);
		statCuda = hipFree(pnInfo_d);
		statCuda = hipFree(pdWork_d);

		return -6;
	}

	statCuda = hipMemcpyAsync(pdX, pdB_d, sizeof(double) * m, hipMemcpyDeviceToHost, stream);
	statCuda = hipStreamSynchronize(stream);

	delete[] pdLU;
	delete[] pnIpiv;

	// free resources
	statCuda = hipFree(pdA_d);
	statCuda = hipFree(pdB_d);
	statCuda = hipFree(pnIpiv_d);
	statCuda = hipFree(pnInfo_d);
	statCuda = hipFree(pdWork_d);

	statCusolver = hipsolverDnDestroy(cusolverH);
	statCuda = hipStreamDestroy(stream);
	statCuda = hipDeviceReset();

	return 0;
}
